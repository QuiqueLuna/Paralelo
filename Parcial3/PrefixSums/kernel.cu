#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

__global__ void prefSum(int* a, int* b, int n) {
    int tid = threadIdx.x + blockIdx.x*blockDim.x;

    if (tid < n) {
        for (int i = tid; i < n; i++) {
            b[i] += a[tid];
            __syncthreads();
        }

    }
}
/*
1 1 1 1 1 1
1 2 2 2 2 2
1 2 3 4 4 4
1 2 3 5

1 1 1 1 1 1
1 2 1 2 1 2



1 2 3 4 5 6
*/

int main() {
    int size = 1024;
    int* host_a, *host_b;
    int* dev_a, * dev_b;
    
    host_a = (int*)malloc(size * sizeof(int));
    host_b = (int*)malloc(size * sizeof(int));
    hipMalloc(&dev_a, size * sizeof(size));
    hipMalloc(&dev_b, size * sizeof(size));

    for (int i = 0; i < size; i++) {
        int r1 = (rand() % (10));
        host_a[i] = 1;
        host_b[i] = 0;
        printf("%d ", host_a[i]);
    }
    printf("\n");

    hipMemcpy(dev_a, host_a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, size * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(32);
    dim3 block(64);
    prefSum << <grid, block >> > (dev_a, dev_b, size);
    hipMemcpy(host_b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);

    printf("RES\n");
    for (int i = 0; i < size; i++) {
        printf("%d ", host_b[i]);
    }
    printf("\n");
    return 0;
}