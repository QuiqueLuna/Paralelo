#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void mult(int *a, int *b, int *c){
    int i = threadIdx.x;
    c[i] = a[i] * b[i];
}

int main()
{
    const int N = 3;
    int size = N * sizeof(N);
    const int a[N] = {1, 0, 1};
    const int b[N] = {2, 4, 3};
    int c[N] = {0, 0, 0};

    int* d_a = 0;
    int* d_b = 0;
    int* d_c = 0;

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
    hipMemcpy(d_c,c,size,hipMemcpyHostToDevice);

    mult << <1, N >> > (d_a, d_b, d_c);
    hipDeviceSynchronize();
    
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
    

    printf("{1,0,1} * {2,4,3} = {%d,%d,%d}\n",c[0], c[1], c[2]);

    hipDeviceReset();

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
