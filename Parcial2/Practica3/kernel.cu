#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>


void bubble_sort(int* a, int n) {
    for (int i = 0; i < n - 1; i++) {
        for (int j = 0; j < n - i - 1; j++) {
            if (a[j] > a[j + 1]) {
                int aux = a[j + 1];
                a[j + 1] = a[j];
                a[j] = aux;
            }
        }
    }
}
__global__ void search(int* a, int n, int* pos, int look) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        if (a[tid] == look) {
            *pos = tid;
        }
    }
}
int main() {
    int size = 32;
    int look = 144;
    int* host_a, * res, *pos;
    int* dev_a, *dev_pos;
    host_a = (int*)malloc(size * sizeof(int));
    pos = (int*)malloc(sizeof(int));
    pos[0] = -1;
    res = (int*)malloc(size * sizeof(int));
    hipMalloc(&dev_a, size * sizeof(int));
    hipMalloc(&dev_pos, sizeof(int));


    for (int i = 0; i < size; i++) {
        int r1 = (rand() % (256));
        host_a[i] = r1;
        printf("%d ", host_a[i]);
    }
    printf("\n");
    
    hipMemcpy(dev_a, host_a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_pos, pos, sizeof(int), hipMemcpyHostToDevice);
    
    dim3 grid(size>=1024 ? size/1024:1);
    dim3 block(1024);
    search << <grid, block >> > (dev_a, size,dev_pos,look);
    hipDeviceSynchronize();
    
    hipMemcpy(pos, dev_pos, sizeof(int), hipMemcpyDeviceToHost);

    if (pos[0] == -1) {
        printf("Not Found\n");
    }
    else {
        printf("Found at %d index\n", pos[0]);
    }

    free(host_a);
    free(pos);
    free(res);
    hipFree(dev_a);
    hipFree(dev_pos);
    return 0;
}